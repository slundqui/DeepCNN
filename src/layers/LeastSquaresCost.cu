#include "hip/hip_runtime.h"
/**
 * LeastSquaresCost.cu
 * Author: Sheng Lundquist
 **/

#include "../utils.hpp"
#include "../cuda_utils.hpp"
#include "../kernels.hpp"


//CUDA kernel
__global__ void k_leastSqTotalCost(float* truth, float* estimate, int batchcount, int bSize, float* out){
   //Linear index into batch and features
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   //If within range
   if(idx < batchcount){
      //Calculate sum of position
      float sumMe = ((float).5/bSize) * pow(truth[idx] - estimate[idx], 2);
      //Atomic add into output
      atomicAdd(out, sumMe);
   }
}

__global__ void k_leastSqCalcGrad(float* truth, float* estimate, int batchcount, int batch, float* out){
   //Linear index into batch and features
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   //If within range
   if(idx < batchcount){
      out[idx] = ((float)1/batch) * (truth[idx] - estimate[idx]);
   }
}

void leastSqTotalCostRunSize(int* gridSize, int* blockSize, int batchcount){
   calcRunSize((void*)&k_leastSqTotalCost, gridSize, blockSize, batchcount);
}

void leastSqCalcGradRunSize(int* gridSize, int* blockSize, int batchcount){
   calcRunSize((void*)&k_leastSqCalcGrad, gridSize, blockSize, batchcount);
}

void leastSqTotalCost(float* truth, float* estimate, int batchcount, int batch, float* out, int gridSize, int blockSize){
   //Reset final cost device variable
   CudaError(hipMemset(out, 0, sizeof(float)));
   CudaError(hipDeviceSynchronize());
   k_leastSqTotalCost<<< gridSize, blockSize>>> (truth, estimate, batchcount, batch, out);
   CudaError(hipDeviceSynchronize());
   CudaCallError();
}

void leastSqCalcGrad(float* truth, float* estimate, int batchcount, int batch, float* out, int gridSize, int blockSize){
   //Reset final cost device variable
   CudaError(hipDeviceSynchronize());
   k_leastSqCalcGrad<<< gridSize, blockSize>>> (truth, estimate, batchcount, batch, out);
   CudaError(hipDeviceSynchronize());
   CudaCallError();
}


