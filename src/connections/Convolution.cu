#include "hip/hip_runtime.h"
/**
 * Convolution.cu
 * Author: Sheng Lundquist
 **/

#include "../utils.hpp"
#include "../cuda_utils.hpp"

extern "C" void convLearningRule(float* d_Weight, float* d_dWeight, float* d_GWeight, int count, float eps, float mom, float decay, int n_blocks, int block_size); 
extern "C" void calcSizeLearn(int* h_block_size, int* h_n_blocks, int count);


//CUDA kernel
__global__ void k_convLearningRule(float* d_Weight, float* d_dWeight, float* d_GWeight, int count, float eps, float mom, float decay){ 
   //Linear index into batch and features
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   //If within range
   if(idx < count){
      d_dWeight[idx] = mom*d_dWeight[idx] - decay*eps*d_Weight[idx] + eps * d_GWeight[idx];
      d_Weight[idx] += d_dWeight[idx];
      //d_Weight[idx] = d_Weight[idx] + (eps * d_GWeight[idx]);
   }
}

void convLearningRule(float* d_Weight, float* d_dWeight, float* d_GWeight, int count, float eps, float mom, float decay, int n_blocks, int block_size){ 
   //Reset final cost device variable
   CudaError(hipDeviceSynchronize());
   k_convLearningRule<<< n_blocks, block_size >>> (d_Weight, d_dWeight, d_GWeight, count, eps, mom, decay);
   CudaCallError();
   CudaError(hipDeviceSynchronize());
}

void calcSizeLearn(int* h_block_size, int* h_n_blocks, int count){
   int minGridSize;
   //Calculate efficient block and grid size
   CudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, h_block_size, &k_convLearningRule, 0, count));
   (*h_n_blocks) = count/(*h_block_size) + (count%(*h_block_size) == 0 ? 0 : 1);
}

